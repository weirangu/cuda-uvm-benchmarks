// Courtesy of https://devblogs.nvidia.com/parallelforall/unified-memory-cuda-beginners/
// REMOVE ME: Uncommnet the code only upon full implementation or get seg-fault
 

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
 
// CUDA kernel to add elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}
 
int main(void)
{
  int N = 1<<20;
  float *x, *y;
 
  // Allocate Unified Memory -- accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));
 
  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipEvent_t start, end;
  float time;

	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);

  // Launch kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  add<<<numBlocks, blockSize>>>(N, x, y);
 
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();


	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);
	fprintf(stdout, "%0.6lf\n", time);
 
  // Free memory
  hipFree(x);
  hipFree(y);
 
  return 0;
}

