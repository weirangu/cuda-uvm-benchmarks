#include "hip/hip_runtime.h"
/**
 * atax.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.5

#define GPU_DEVICE 0

/* Problem size. */
#define NX 4096
#define NY 4096

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 256
#define DIM_THREAD_BLOCK_Y 1

#ifndef M_PI
#define M_PI 3.14159
#endif

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;



void init_array(DATA_TYPE *x, DATA_TYPE *A)
{
	int i, j;

	for (i = 0; i < NX; i++)
	{
		x[i] = i * M_PI;
		for (j = 0; j < NY; j++)
		{
			A[i*NY + j] = ((DATA_TYPE) i*(j)) / NX;
		}
	}
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void atax_kernel1(DATA_TYPE *A, DATA_TYPE *x, DATA_TYPE *tmp)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < NX)
	{
		int j;
		for(j=0; j < NY; j++)
		{
			tmp[i] += A[i * NY + j] * x[j];
		}
	}
}

__global__ void atax_kernel2(DATA_TYPE *A, DATA_TYPE *y, DATA_TYPE *tmp)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (j < NY)
	{
		int i;
		for(i=0; i < NX; i++)
		{
			y[j] += A[i * NY + j] * tmp[i];
		}
	}
}


void ataxGpu(DATA_TYPE* A_gpu, DATA_TYPE* x_gpu, DATA_TYPE* y_gpu, DATA_TYPE* tmp_gpu, DATA_TYPE* y_outputFromGpu)
{
	double t_start, t_end;
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid1((size_t)(ceil( ((float)NX) / ((float)block.x) )), 1);
	dim3 grid2((size_t)(ceil( ((float)NY) / ((float)block.x) )), 1);

	t_start = rtclock();
	atax_kernel1<<< grid1, block >>>(A_gpu,x_gpu,tmp_gpu);
  hipDeviceSynchronize();
	atax_kernel2<<< grid2, block >>>(A_gpu,y_gpu,tmp_gpu);
  hipDeviceSynchronize();
	t_end = rtclock();
	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);
	
	hipMemcpy(y_outputFromGpu, y_gpu, sizeof(DATA_TYPE) * NX, hipMemcpyDeviceToHost);
}


int main(int argc, char** argv)
{
	DATA_TYPE* A;
	DATA_TYPE* x;
	DATA_TYPE* y;
	DATA_TYPE* y_outputFromGpu;
	DATA_TYPE* tmp;

	hipMallocManaged(&A, NX*NY*sizeof(DATA_TYPE));
	hipMallocManaged(&x, NY*sizeof(DATA_TYPE));
	hipMallocManaged(&y, NY*sizeof(DATA_TYPE));
	y_outputFromGpu = (DATA_TYPE*)malloc(NY*sizeof(DATA_TYPE));
	tmp = (DATA_TYPE*)malloc(NX*sizeof(DATA_TYPE));

	init_array(x, A);

	GPU_argv_init();
	ataxGpu(A, x, y, tmp, y_outputFromGpu);
	

	hipFree(A);
	hipFree(x);
	hipFree(y);
	free(y_outputFromGpu);
	free(tmp);

  	return 0;
}

