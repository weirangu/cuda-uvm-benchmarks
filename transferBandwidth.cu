#define PG (4*1024)

#include <hip/hip_runtime.h>
#include <stdio.h>

int main(void)
{
  int N = 2044*1024;
  float *x, *d_x;

  x = (float*)malloc(N*sizeof(float));
 
  hipMalloc(&d_x, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 0;
  }


	hipEvent_t start, end;
	float time;
  int current = 0;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start);
  for (int i = 0; i < 9; i++) { 
    hipMemcpy((d_x+current), (x+current), (int)(1024*pow(2.0,(i+2))), hipMemcpyHostToDevice);
    current += (int)(1024*pow(2.0,(i+2)));
  }
  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&time, start, end);

  printf("time: %f\n", time);
  
  // Free memory
  hipFree(d_x);
  free(x);

  return 0;
}

