// Courtesy of https://devblogs.nvidia.com/parallelforall/easy-introduction-cuda-c-and-c/ 


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
 
// CUDA kernel to add elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}
 
int main(int argc, char** argv)
{

  if(argc < 2)
    return 1;

  int N = 1<<atoi(argv[1]);
  float *x, *y, *d_x, *d_y;

  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
 
  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }


  hipEvent_t start, end;
  float time;
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;


  //for(int i = 0; i < 5; i++){


    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

   
    add<<<numBlocks, blockSize>>>(N, d_x, d_y);
    hipDeviceSynchronize();

    hipEventRecord(end);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    fprintf(stdout, "%0.6lf\n", time);
  //}

  // Free memory
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
  
  return 0;
}

